
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__
void convolution(int n, int m, short int *mask, unsigned char *original, unsigned char *resultado){  
    int i = blockIdx.y * blockDim.y + threadIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.y;
    int aux_i, aux_j;
    int aux;
    int pixel_resultante;
    int p, q; // for interno, da mascara 3x3

    if (i >= n || j >= m){
        return;
    }

    pixel_resultante = 0;
    aux_i = i; 
    aux_j = j;
    aux = j;
    for(p = 0; p < 3; p++){
        for(q = 0; q < 3; q++){
            pixel_resultante += original[aux_i*n + aux_j] * mask[p*3 + q];
            aux_j++;
        }
        j = aux;
        aux_i++;
    }
    //por estarmos utilizando uma matriz 3x3 de gauss, após a soma das multiplicações devemos dividir por 16
    resultado[i*n + j] = pixel_resultante/16;
}

int main(int argc, char **argv){
    FILE *imagem;
    FILE *nova_imagem;
    char *nome_imagem;
    char *nome_imagem_saida;
    char key[128];
    int i, j, m, n, max;


    if (argc != 3){
        printf("Erro, o programa deve receber o nome da imagem de entrada \n"
                "e um nome para imagem de saida.");
        return 0;
    }

    nome_imagem = argv[1];
    nome_imagem_saida = argv[2];

    imagem = fopen(nome_imagem , "r") ; // Abre o arquivo no modo leitura
    if(imagem == NULL){ // Verificase o arquivo existe e foi aberto
        printf("Erro na abertura do arquivo %s\n", nome_imagem);
        return 0;
    }

    // Le cabecalho
    fscanf(imagem, "%s", key);

    // Imagens PGM tem "P5" na primeira linha
    if(strcmp(key,"P5") != 0){
        printf("Arquivo precisa ser PGM!\n") ;
        fclose(imagem);
        return 0;
    }

    //Próximos valores do cabeçalho após P5 são: numero de colunas, numero de linhas, e valor máximo.
    fscanf(imagem, "%d %d %d", &m, &n, &max) ;

    // Matriz para guardar a imagem original
    // Utilizando o método de bordas "pretas"
    unsigned char *original, *d_original;
    int n_col = m+2;
    int n_row = n+2;

    hipMalloc(&d_original, n_row * n_col * sizeof(unsigned char*));
    original = (unsigned char*) malloc(n_row * n_col * sizeof(unsigned char*));


    // Matriz pra servir de buffer pra imagem resultado
    unsigned char *resultado, *d_resultado;
    resultado = (unsigned char*) malloc(n * m *sizeof(unsigned char*));
    hipMalloc(&d_resultado, n_row * n_col * sizeof(unsigned char*));


    // Leitura da imagem original
    for(i = 1; i < n+1; i++){
        for(j = 1; j < m+1; j++){
            fscanf(imagem, "%c", &original[i*n + j]);
        }
    }

    hipMemcpy(d_original, original, n * m *sizeof(unsigned char*), hipMemcpyHostToDevice);

    // matriz de convolução gaussiana
    short int mask[9] = {1, 2, 1, 2, 4, 2, 1, 2, 1};


    short int *d_mask;
    hipMalloc(&d_mask, 3 * 3 * sizeof(short int*));
    hipMemcpy(d_mask, mask, 3 * 3 * sizeof(short int*), hipMemcpyHostToDevice);

    dim3 grid(32,32);
    dim3 block(32,32);
    convolution<<<grid,block>>>(n, m, d_mask, d_original, d_resultado);
   
    hipMemcpy(resultado, d_resultado, n * m *sizeof(unsigned char*), hipMemcpyDeviceToHost);
    printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
    hipDeviceSynchronize();


    // abrir nova imagem em modo de escrita e "copiar" o cabeçalho da imagem original
    nova_imagem = fopen(nome_imagem_saida , "w");
    fprintf(nova_imagem,"P5\n%d %d\n %d\n", m, n, max);

    // escrever no arquivo resultado
    for(i = 0; i < n; i++){
        for(j = 0; j < m; j++){
            fprintf(nova_imagem, "%c", (char) resultado[i*n + j]);
        }
    }


    fclose(imagem);
    fclose(nova_imagem);

    free(original);
    free(resultado);

    hipFree(d_mask);
    hipFree(d_original);
    hipFree(d_resultado);

    return 0;
}